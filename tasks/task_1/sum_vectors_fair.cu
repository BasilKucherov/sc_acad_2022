
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <pthread.h> 

#define ARR_SIZE (104857600) //  (4294967296)
#define ARR_LEN (ARR_SIZE/sizeof(double))
#define THREAD_NUMBER 4


struct thread_data {
  size_t start;
  size_t end;
  double* arr;
};


__global__ void do_math_gpu (double* dC, size_t arr_len) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  
  if(i < arr_len) {
    double sum = 0.;
    double ab = sin((double)i) * cos(2*(double)i-5);

    for(int j = 0; j < 100; j++) {
        sum += sin(ab + (double)j);
    }
    
    dC[i] = sum;
  }
}


void* job(void* arg) {
  struct thread_data* p = (struct thread_data*)arg;

    double sum = 0;
    double ab = 0;

    size_t start = p->start;
    size_t end = p->end;

    for(size_t i = start; i < end; i++) {
        sum = 0;
        ab = sin((double)i) * cos(2*(double)i-5);

        for(int j = 0; j < 100; j++) {
            sum += sin(ab + j);
        }

        (p->arr)[i] = sum;
    }

    return 0;
}


void do_math_cpu(double* C, size_t arr_len) {
    pthread_t thrds[THREAD_NUMBER];
    struct thread_data data[THREAD_NUMBER];
    size_t step = (arr_len + THREAD_NUMBER - 1) / THREAD_NUMBER;
    size_t start = 0;
    size_t end = 0;

    for (int i = 0; i < THREAD_NUMBER; i++) {
        
        data[i].start = start;

        if (i == THREAD_NUMBER - 1) {
            end = arr_len;
        } else {
            end += step;
        }

        data[i].end = end;
        data[i].arr = C;

        printf("Thread #%d:\n", i);
        printf("\tStart: %lu\n", data[i].start);
        printf("\tEnd: %lu\n", data[i].end);

        start = end;



        pthread_create(thrds + i, NULL, &job, (void*)(data + i));
    }

  for (int i = 0; i < THREAD_NUMBER; i++) {
    pthread_join(thrds[i], NULL);
  }
}


void compare_arrs(double* arr_1, double* arr_2, size_t arr_len)
{
    double max_err = 0.;
    size_t max_err_idx = 0;
    double avg_err = 0.;
    double accum_err = 0.;

    double diff = 0.;

    for(size_t i = 0; i < arr_len; i++)
    {
        diff = fabs(arr_1[i] - arr_2[i]);

        if (diff > max_err) {
            max_err = diff;
            max_err_idx = i;
        }

        accum_err += diff;
     }

    avg_err = accum_err / arr_len;

    printf("Max error idx = %lu: %lf -- %lf\n", max_err_idx, arr_1[max_err_idx],  arr_2[max_err_idx]);
    printf("Max error = %.16lf\n", max_err);
    printf("Avg error = %.16lf\n", avg_err);
}


int main(void)
{
    printf("Arr size: %lu\n", ARR_SIZE);
    printf("Arr len: %lu\n\n", ARR_LEN);
    float cudamemcpy_dth_time = 0.;
    float calc_cpu_time = 0., calc_gpu_time = 0.;

    hipEvent_t start, stop; 
    hipEventCreate ( &start ); 
    hipEventCreate ( &stop );

    double* hdC = (double*) malloc(ARR_SIZE);
    double *dC;
    hipMalloc ((void**) &dC, ARR_SIZE);

    hipEventRecord ( start, 0);
    do_math_gpu<<<(ARR_LEN + 1023) / 1024, 1024>>>(dC, ARR_LEN);
    hipEventRecord ( stop, 0 );

    hipEventSynchronize ( stop );
    hipEventElapsedTime ( &calc_gpu_time, start, stop );

    hipEventRecord ( start, 0);

    hipMemcpy ( hdC, dC, ARR_SIZE, hipMemcpyDeviceToHost );
    
    hipEventRecord ( stop, 0 );
    hipEventSynchronize ( stop );
    hipEventElapsedTime ( &cudamemcpy_dth_time, start, stop );

    double* hC = (double*) malloc(ARR_SIZE);

    hipEventRecord ( start, 0);

    do_math_cpu(hC, ARR_LEN);

    hipEventRecord ( stop, 0 );
    hipEventSynchronize ( stop );
    hipEventElapsedTime ( &calc_cpu_time, start, stop );

    compare_arrs(hC, hdC, ARR_LEN);

    printf ("\nGPU D2H memory copy time: %f ms\n", cudamemcpy_dth_time );
    printf ("GPU calculation time: %f ms\n\n", calc_gpu_time );
    printf ("GPU total time: %f ms\n",  cudamemcpy_dth_time + calc_gpu_time);

    printf ("CPU calculation time: %f ms\n", calc_cpu_time );

    printf("Acceleration (only calculation): %.1lfx (%.1lfx)\n", calc_cpu_time / (cudamemcpy_dth_time + calc_gpu_time), calc_cpu_time / calc_gpu_time);

    free(hC);
    free(hdC);
    hipFree(dC);
    hipEventDestroy ( start );
    hipEventDestroy ( stop );
    return 0;
}
